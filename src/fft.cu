#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, The Bifrost Authors. All rights reserved.
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * * Redistributions of source code must retain the above copyright
 *   notice, this list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimer in the
 *   documentation and/or other materials provided with the distribution.
 * * Neither the name of The Bifrost Authors nor the names of its
 *   contributors may be used to endorse or promote products derived
 *   from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*! \file fft.cu
 *  \brief This file wraps CUFFT functionality into the Bifrost C++ API.
 */

/*
  TODO: Implicitly padded/cropped transforms using load callback
        Optional fftshift of output
          Cyclic shift along each axis of n//2 elements
            (or negative shift for ifftshift).
*/

#include <bifrost/fft.h>
#include "assert.hpp"
#include "utils.hpp"
#include "cuda.hpp"
#include "ShapeIndexer.cuh"
#include "ArrayIndexer.cuh"
#include <thrust/device_vector.h>

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

const char* _cufftGetErrorString(hipfftResult status) {
#define DEFINE_CUFFT_RESULT_CASE(x) case x: return #x
	switch( status ) {
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_SUCCESS);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_INVALID_PLAN);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_ALLOC_FAILED);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_INVALID_TYPE);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_INVALID_VALUE);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_INTERNAL_ERROR);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_EXEC_FAILED);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_SETUP_FAILED);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_INVALID_SIZE);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_UNALIGNED_DATA);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_INCOMPLETE_PARAMETER_LIST);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_INVALID_DEVICE);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_PARSE_ERROR);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_NO_WORKSPACE);
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_NOT_IMPLEMENTED);
	DEFINE_CUFFT_RESULT_CASE(CUFFT_LICENSE_ERROR);
#if CUDA_VERSION >= 7500
	DEFINE_CUFFT_RESULT_CASE(HIPFFT_NOT_SUPPORTED);
#endif
	default: return "Unknown CUBLAS error";
	}
#undef DEFINE_CUFFT_RESULT_CASE
}

BFstatus bifrost_status(hipfftResult status) {
	switch(status) {
	case HIPFFT_SUCCESS:          return BF_STATUS_SUCCESS;
	case HIPFFT_ALLOC_FAILED:     return BF_STATUS_MEM_ALLOC_FAILED;
	case HIPFFT_EXEC_FAILED:      return BF_STATUS_DEVICE_ERROR;
	case HIPFFT_NOT_IMPLEMENTED:  return BF_STATUS_UNSUPPORTED;
#if CUDA_VERSION >= 7500
	case HIPFFT_NOT_SUPPORTED:    return BF_STATUS_UNSUPPORTED;
#endif
	default: return BF_STATUS_INTERNAL_ERROR;
    }
}

#define BF_CHECK_CUFFT_EXCEPTION(call) \
	do { \
		hipfftResult cufft_ret = call; \
		if( cufft_ret != HIPFFT_SUCCESS ) { \
			BF_DEBUG_PRINT(_cufftGetErrorString(cufft_ret)); \
		} \
		BF_ASSERT_EXCEPTION(cufft_ret == HIPFFT_SUCCESS, \
		                    bifrost_status(cufft_ret)); \
	} while(0)

#define BF_CHECK_CUFFT(call) \
	do { \
		hipfftResult cufft_ret = call; \
		if( cufft_ret != HIPFFT_SUCCESS ) { \
			BF_DEBUG_PRINT(_cufftGetErrorString(cufft_ret)); \
		} \
		BF_ASSERT(cufft_ret == HIPFFT_SUCCESS, \
		          bifrost_status(cufft_ret)); \
	} while(0)

struct CallbackData {
	int ptr_offset;
};

__device__
hipfftComplex callback_load_ci4(void*  dataIn,
                               size_t offset,
                               void*  callerInfo,
                               void*  sharedPointer) {
	// WAR for CUFFT insisting on pointers aligned to sizeof(hipfftComplex)
	CallbackData* callback_data = (CallbackData*)callerInfo;
	*(char*)&dataIn += callback_data->ptr_offset;
	
	int8_t packed = ((int8_t*)dataIn)[offset];
	int8_t real = packed & 0xF0;
	int8_t imag = packed << 4;
	return make_float2(real * (1.f/128),
	                   imag * (1.f/128));
}
__device__
hipfftComplex callback_load_ci8(void*  dataIn,
                               size_t offset,
                               void*  callerInfo,
                               void*  sharedPointer) {
	// WAR for CUFFT insisting on pointers aligned to sizeof(hipfftComplex)
	CallbackData* callback_data = (CallbackData*)callerInfo;
	*(char*)&dataIn += callback_data->ptr_offset;
	
	char2 val = ((char2*)dataIn)[offset];
	return make_float2(val.x * (1.f/128),
	                   val.y * (1.f/128));
}
__device__
hipfftComplex callback_load_ci16(void*  dataIn,
                                size_t offset,
                                void*  callerInfo,
                                void*  sharedPointer) {
	// WAR for CUFFT insisting on pointers aligned to sizeof(hipfftComplex)
	CallbackData* callback_data = (CallbackData*)callerInfo;
	*(char*)&dataIn += callback_data->ptr_offset;
	
	short2 val = ((short2*)dataIn)[offset];
	return make_float2(val.x * (1.f/32768),
	                   val.y * (1.f/32768));
}
static __device__ hipfftCallbackLoadC callback_load_ci4_dptr  = callback_load_ci4;
static __device__ hipfftCallbackLoadC callback_load_ci8_dptr  = callback_load_ci8;
static __device__ hipfftCallbackLoadC callback_load_ci16_dptr = callback_load_ci16;

template<typename T>
struct is_signed { enum { value = (((T)(-1)) < 0) }; };

template<typename T>
__host__ __device__
inline T maxval(T x=T()) { return (1<<(sizeof(T)*8-is_signed<T>::value)) - 1; }

template<typename T>
__device__
hipfftReal callback_load_real(void*  dataIn,
                             size_t offset,
                             void*  callerInfo,
                             void*  sharedPointer) {
	// WAR for CUFFT insisting on pointers aligned to sizeof(hipfftComplex)
	CallbackData* callback_data = (CallbackData*)callerInfo;
	*(char*)&dataIn += callback_data->ptr_offset;
	
	T val = ((T*)dataIn)[offset];
	return val * (1.f/(maxval<T>()+1));
}
static __device__ hipfftCallbackLoadR callback_load_i8_dptr  = callback_load_real<int8_t>;
static __device__ hipfftCallbackLoadR callback_load_i16_dptr = callback_load_real<int16_t>;
static __device__ hipfftCallbackLoadR callback_load_u8_dptr  = callback_load_real<uint8_t>;
static __device__ hipfftCallbackLoadR callback_load_u16_dptr = callback_load_real<uint16_t>;

class BFfft_impl {
	hipfftHandle      _handle;
	bool             _real_in;
	bool             _real_out;
	int              _nbit;
	BFdtype          _itype;
	BFdtype          _otype;
	int              _batch_shape[BF_MAX_DIMS];
	size_t           _workspace_size;
	thrust::device_vector<char> _dv_tmp_storage;
	thrust::device_vector<CallbackData> _dv_callback_data;
	
	BFstatus execute_impl(void*   idata,
	                      BFdtype itype,
	                      void*   odata,
	                      BFdtype otype,
	                      BFbool  inverse,
	                      void*   tmp_storage,
	                      size_t  tmp_storage_size);
	// No copy-assign
	BFfft_impl(BFfft_impl const& );
	BFfft_impl& operator=(BFfft_impl const& );
public:
	BFfft_impl();
	~BFfft_impl();
	BFstatus init(BFarray const* in,
	              BFarray const* out,
	              int            rank,
	              int     const* axes,
	              size_t*        tmp_storage_size);
	BFstatus execute(BFarray const* in,
	                 BFarray const* out,
	                 BFbool         inverse,
	                 void*          tmp_storage,
	                 size_t         tmp_storage_size);
};

BFfft_impl::BFfft_impl() {
	BF_CHECK_CUFFT_EXCEPTION( hipfftCreate(&_handle) );
}
BFfft_impl::~BFfft_impl() {
	hipfftDestroy(_handle);
}

BFstatus BFfft_impl::init(BFarray const* in,
                          BFarray const* out,
                          int            rank,
                          int     const* axes,
                          size_t*        tmp_storage_size) {
	BF_ASSERT(rank > 0 && rank <= BF_MAX_DIMS, BF_STATUS_INVALID_ARGUMENT);
	BF_ASSERT(rank <= in->ndim, BF_STATUS_INVALID_ARGUMENT);
	//BF_ASSERT(
	// TODO: More assertions...
	
	_real_in  = !BF_DTYPE_IS_COMPLEX( in->dtype);
	_real_out = !BF_DTYPE_IS_COMPLEX(out->dtype);
	
	int mutable_axes[BF_MAX_DIMS];
	for( int d=0; d<rank; ++d ) {
		// Default to last 'rank' axes
		mutable_axes[d] = axes ? axes[d] : in->ndim-rank+d;
		// Allow negative axis numbers
		if( mutable_axes[d] < 0 ) {
			mutable_axes[d] += in->ndim;
		}
	}
	axes = mutable_axes;
	for( int d=0; d<in->ndim; ++d ) {
		long ilength =  in->shape[d];
		long olength = out->shape[d];
		if( (!_real_in && !_real_out) ||
		    d != axes[rank-1] ) {
			BF_ASSERT(ilength == olength,
			          BF_STATUS_INVALID_SHAPE);
		} else if( !_real_out ) {
			// Special case for last dim of R2C transforms
			BF_ASSERT(olength == ilength/2+1,
			          BF_STATUS_INVALID_SHAPE);
		} else {
			// Special case for last dim of C2R transforms
			BF_ASSERT(ilength == olength/2+1,
			          BF_STATUS_INVALID_SHAPE);
		}
		// Initialize batch shape to data shape
		_batch_shape[d] = _real_in ? ilength : olength;
	}
	// Compute transform shape and strides
#if CUDA_VERSION >= 7500
	typedef long long int_array_type;
#else
	typedef int int_array_type;
#endif
	int_array_type   shape[BF_MAX_DIMS];
	int_array_type inembed[BF_MAX_DIMS];
	int_array_type onembed[BF_MAX_DIMS];
	for( int d=0; d<rank; ++d ) {
		long ilength =  in->shape[axes[d]];
		long olength = out->shape[axes[d]];
		shape[d] = _real_in ? ilength : olength;
		if( d > 0 ) {
			BF_ASSERT( in->strides[axes[d-1]] %  in->strides[axes[d]] == 0,
			                     BF_STATUS_UNSUPPORTED_STRIDE);
			BF_ASSERT(out->strides[axes[d-1]] % out->strides[axes[d]] == 0,
			                    BF_STATUS_UNSUPPORTED_STRIDE);
			// Note: These implicitly span the batch dims where necessary
			inembed[d] =  in->strides[axes[d-1]] /  in->strides[axes[d]];
			onembed[d] = out->strides[axes[d-1]] / out->strides[axes[d]];
		} else {
			inembed[d] =  in->shape[axes[d]];
			onembed[d] = out->shape[axes[d]];
		}
		// This is not a batch dim, so exclude it from _batch_shape
		_batch_shape[axes[d]] = 1;
	}
	int itype_nbyte = BF_DTYPE_NBYTE( in->dtype);
	int otype_nbyte = BF_DTYPE_NBYTE(out->dtype);
	int istride_bytes = in->strides[axes[rank-1]];
	BF_ASSERT(istride_bytes % itype_nbyte == 0,
	                    BF_STATUS_UNSUPPORTED_STRIDE);
	int istride = istride_bytes / itype_nbyte;
	int ostride_bytes = out->strides[axes[rank-1]];
	BF_ASSERT(ostride_bytes % otype_nbyte == 0,
	                    BF_STATUS_UNSUPPORTED_STRIDE);
	int ostride = ostride_bytes / otype_nbyte;
	
	// Use longest batch dim as cuFFT batch parameter
	int batch_dim;
	bool fastest_dim_is_batch_dim = axes[rank-1] != in->ndim-1;
	if( (_real_in || _real_out) && fastest_dim_is_batch_dim ) {
		// Set the inner dim as the kernel batch, as a WAR for CUFFT requiring
		//   complex-aligned memory.
		batch_dim = in->ndim-1;
	} else {
		// Otherwise use the largest batch dim as the kernel batch for best
		//   performance.
		batch_dim = argmax_last(_batch_shape, in->ndim);
	}
	long batch = _batch_shape[batch_dim];
	_batch_shape[batch_dim] = 1;
	long idist =  in->strides[batch_dim] / itype_nbyte;
	long odist = out->strides[batch_dim] / otype_nbyte;
	
	bool fp64 = (out->dtype == BF_DTYPE_F64 ||
	             out->dtype == BF_DTYPE_CF64);
	_nbit = fp64 ? 64 : 32;
	_itype =  in->dtype;
	_otype = out->dtype;
	hipfftType type;
	if(      !_real_in && !_real_out ) { type = fp64 ? HIPFFT_Z2Z : HIPFFT_C2C; }
	else if(  _real_in && !_real_out ) { type = fp64 ? HIPFFT_D2Z : HIPFFT_R2C; }
	else if( !_real_in &&  _real_out ) { type = fp64 ? HIPFFT_Z2D : HIPFFT_C2R; }
	else {
		BF_FAIL("Complex input and/or output",
		        BF_STATUS_INVALID_DTYPE);
	}
	BF_CHECK_CUFFT( hipfftSetAutoAllocation(_handle, false) );
#if CUDA_VERSION >= 7500
	BF_CHECK_CUFFT( hipfftMakePlanMany64(_handle,
#else
	BF_CHECK_CUFFT( hipfftMakePlanMany  (_handle,
#endif
	                                    rank, shape,
	                                    inembed, istride, idist,
	                                    onembed, ostride, odist,
	                                    type,
	                                    batch,
	                                    &_workspace_size) );
	
	hipfftCallbackLoadC callback_load_c_hptr;
	hipfftCallbackLoadR callback_load_r_hptr;
	_dv_callback_data.resize(1);
	CallbackData* callback_data = thrust::raw_pointer_cast(&_dv_callback_data[0]);
	// TODO: Try to reduce repetition here
	switch( in->dtype ) {
	case BF_DTYPE_CI4: {
		BF_CHECK_CUDA( hipMemcpyFromSymbol(&callback_load_c_hptr,
		                                    HIP_SYMBOL(callback_load_ci4_dptr),
		                                    sizeof(hipfftCallbackLoadC)),
		               BF_STATUS_DEVICE_ERROR );
		BF_CHECK_CUFFT( hipfftXtSetCallback(_handle,
		                                   (void**)&callback_load_c_hptr,
		                                   HIPFFT_CB_LD_COMPLEX,
		                                   (void**)&callback_data) );
		break;
	}
	case BF_DTYPE_CI8: {
		BF_CHECK_CUDA( hipMemcpyFromSymbol(&callback_load_c_hptr,
		                                    HIP_SYMBOL(callback_load_ci8_dptr),
		                                    sizeof(hipfftCallbackLoadC)),
		               BF_STATUS_DEVICE_ERROR );
		BF_CHECK_CUFFT( hipfftXtSetCallback(_handle,
		                                   (void**)&callback_load_c_hptr,
		                                   HIPFFT_CB_LD_COMPLEX,
		                                   (void**)&callback_data) );
		break;
	}
	case BF_DTYPE_CI16: {
		BF_CHECK_CUDA( hipMemcpyFromSymbol(&callback_load_c_hptr,
		                                    HIP_SYMBOL(callback_load_ci16_dptr),
		                                    sizeof(hipfftCallbackLoadC)),
		               BF_STATUS_DEVICE_ERROR );
		BF_CHECK_CUFFT( hipfftXtSetCallback(_handle,
		                                   (void**)&callback_load_c_hptr,
		                                   HIPFFT_CB_LD_COMPLEX,
		                                   (void**)&callback_data) );
		break;
	}
	case BF_DTYPE_I8: {
		BF_CHECK_CUDA( hipMemcpyFromSymbol(&callback_load_r_hptr,
		                                    HIP_SYMBOL(callback_load_i8_dptr),
		                                    sizeof(hipfftCallbackLoadR)),
		               BF_STATUS_DEVICE_ERROR );
		BF_CHECK_CUFFT( hipfftXtSetCallback(_handle,
		                                   (void**)&callback_load_r_hptr,
		                                   HIPFFT_CB_LD_REAL,
		                                   (void**)&callback_data) );
		break;
	}
	case BF_DTYPE_I16: {
		BF_CHECK_CUDA( hipMemcpyFromSymbol(&callback_load_r_hptr,
		                                    HIP_SYMBOL(callback_load_i16_dptr),
		                                    sizeof(hipfftCallbackLoadR)),
		               BF_STATUS_DEVICE_ERROR );
		BF_CHECK_CUFFT( hipfftXtSetCallback(_handle,
		                                   (void**)&callback_load_r_hptr,
		                                   HIPFFT_CB_LD_REAL,
		                                   (void**)&callback_data) );
		break;
	}
	case BF_DTYPE_U8: {
		BF_CHECK_CUDA( hipMemcpyFromSymbol(&callback_load_r_hptr,
		                                    HIP_SYMBOL(callback_load_u8_dptr),
		                                    sizeof(hipfftCallbackLoadR)),
		               BF_STATUS_DEVICE_ERROR );
		BF_CHECK_CUFFT( hipfftXtSetCallback(_handle,
		                                   (void**)&callback_load_r_hptr,
		                                   HIPFFT_CB_LD_REAL,
		                                   (void**)&callback_data) );
		break;
	}
	case BF_DTYPE_U16: {
		BF_CHECK_CUDA( hipMemcpyFromSymbol(&callback_load_r_hptr,
		                                    HIP_SYMBOL(callback_load_u16_dptr),
		                                    sizeof(hipfftCallbackLoadR)),
		               BF_STATUS_DEVICE_ERROR );
		BF_CHECK_CUFFT( hipfftXtSetCallback(_handle,
		                                   (void**)&callback_load_r_hptr,
		                                   HIPFFT_CB_LD_REAL,
		                                   (void**)&callback_data) );
		break;
	}
	case BF_DTYPE_CF32: // Fall-through
	case BF_DTYPE_F32:  {
		BF_ASSERT(_nbit == 32, BF_STATUS_INVALID_DTYPE);
		break;
	}
	case BF_DTYPE_CF64: // Fall-through
	case BF_DTYPE_F64: {
		BF_ASSERT(_nbit == 64, BF_STATUS_INVALID_DTYPE);
		break;
	}
	default: {
		BF_FAIL("Supported input data type", BF_STATUS_INVALID_DTYPE);
	}
	}
	
	if( tmp_storage_size ) {
		*tmp_storage_size = _workspace_size;
	}
	return BF_STATUS_SUCCESS;
}

BFstatus BFfft_impl::execute_impl(void*   idata,
                                  BFdtype itype,
                                  void*   odata,
                                  BFdtype otype,
                                  BFbool  inverse,
                                  void*   tmp_storage,
                                  size_t  tmp_storage_size) {
	BF_ASSERT(itype == _itype, BF_STATUS_INVALID_DTYPE);
	BF_ASSERT(otype == _otype, BF_STATUS_INVALID_DTYPE);
	if( !tmp_storage ) {
		BF_TRY(_dv_tmp_storage.resize(_workspace_size));
		tmp_storage = thrust::raw_pointer_cast(&_dv_tmp_storage[0]);
	} else {
		BF_ASSERT(tmp_storage_size >= _workspace_size,
		          BF_STATUS_INSUFFICIENT_STORAGE);
	}
	BF_CHECK_CUFFT( hipfftSetWorkArea(_handle, tmp_storage) );
	
	CallbackData h_callback_data;
	// WAR for CUFFT insisting that pointer be aligned to sizeof(hipfftComplex)
	int alignment = (_nbit == 32 ?
	                 sizeof(hipfftComplex) :
	                 sizeof(hipfftDoubleComplex));
	h_callback_data.ptr_offset = (uintptr_t)idata % sizeof(hipfftComplex);
	*(char**)&idata -= h_callback_data.ptr_offset;
	CallbackData* d_callback_data = thrust::raw_pointer_cast(&_dv_callback_data[0]);
	hipMemcpyAsync(d_callback_data, &h_callback_data, sizeof(CallbackData),
	                hipMemcpyHostToDevice, g_cuda_stream);
	
	BF_ASSERT((uintptr_t)idata % alignment == 0, BF_STATUS_UNSUPPORTED_STRIDE);
	BF_ASSERT((uintptr_t)odata % alignment == 0, BF_STATUS_UNSUPPORTED_STRIDE);
	
	if( !_real_in && !_real_out ) {
		int direction = inverse ? HIPFFT_BACKWARD : HIPFFT_FORWARD;
		if( _nbit == 32 ) {
			BF_CHECK_CUFFT( hipfftExecC2C(_handle, (hipfftComplex*)idata, (hipfftComplex*)odata, direction) );
		} else if( _nbit == 64 ) {
			BF_CHECK_CUFFT( hipfftExecZ2Z(_handle, (hipfftDoubleComplex*)idata, (hipfftDoubleComplex*)odata, direction) );
		} else {
			BF_FAIL("Supported data types", BF_STATUS_UNSUPPORTED_DTYPE);
		}
	} else if( _real_in && !_real_out ) {
		if( _nbit == 32 ) {
			BF_CHECK_CUFFT( hipfftExecR2C(_handle, (hipfftReal*)idata, (hipfftComplex*)odata) );
		} else if( _nbit == 64 ) {
			BF_CHECK_CUFFT( hipfftExecD2Z(_handle, (hipfftDoubleReal*)idata, (hipfftDoubleComplex*)odata) );
		} else {
			BF_FAIL("Supported data types", BF_STATUS_UNSUPPORTED_DTYPE);
		}
	} else if( !_real_in && _real_out ) {
		if( _nbit == 32 ) {
			BF_CHECK_CUFFT( hipfftExecC2R(_handle, (hipfftComplex*)idata, (hipfftReal*)odata) );
		} else if( _nbit == 64 ) {
			BF_CHECK_CUFFT( hipfftExecZ2D(_handle, (hipfftDoubleComplex*)idata, (hipfftDoubleReal*)odata) );
		} else {
			BF_FAIL("Supported data types", BF_STATUS_UNSUPPORTED_DTYPE);
		}
	} else {
		BF_FAIL("Valid data types", BF_STATUS_INVALID_DTYPE);
	}
	return BF_STATUS_SUCCESS;
}

BFstatus BFfft_impl::execute(BFarray const* in,
                             BFarray const* out,
                             BFbool         inverse,
                             void*          tmp_storage,
                             size_t         tmp_storage_size) {
	BF_ASSERT(space_accessible_from( in->space, BF_SPACE_CUDA), BF_STATUS_UNSUPPORTED_SPACE);
	BF_ASSERT(space_accessible_from(out->space, BF_SPACE_CUDA), BF_STATUS_UNSUPPORTED_SPACE);
	// TODO: More assertions
	
	hipStream_t stream = g_cuda_stream;
	// Note: It appears that all transforms from the same plan must be executed
	//         on the same stream to avoid race conditions (use of workspace?).
	BF_CHECK_CUFFT( hipfftSetStream(_handle, stream) );
	ShapeIndexer<BF_MAX_DIMS> shape_indexer(_batch_shape, in->ndim);
	for( long i=0; i<shape_indexer.size(); ++i ) {
		auto inds = shape_indexer.at(i);
		void* idata = array_get_pointer( in, inds);
		void* odata = array_get_pointer(out, inds);
		BFstatus ret = this->execute_impl(idata, in->dtype,
		                                  odata, out->dtype,
		                                  inverse,
		                                  tmp_storage, tmp_storage_size);
		if( ret != BF_STATUS_SUCCESS ) {
			return ret;
		}
	}
	return BF_STATUS_SUCCESS;
}

BFstatus bfFftCreate(BFfft* plan_ptr) {
	BF_ASSERT(plan_ptr, BF_STATUS_INVALID_POINTER);
	BF_TRY_RETURN_ELSE(*plan_ptr = new BFfft_impl(),
	                   *plan_ptr = 0);
}
BFstatus bfFftInit(BFfft          plan,
                   BFarray const* in,
                   BFarray const* out,
                   int            rank,
                   int     const* axes,
                   size_t*        tmp_storage_size) {
	BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
	BF_ASSERT(in,   BF_STATUS_INVALID_POINTER);
	BF_ASSERT(out,  BF_STATUS_INVALID_POINTER);
	return plan->init(in, out, rank, axes, tmp_storage_size);
}
// in, out = complex, complex => [i]fft
// in, out = real, complex    => rfft
// in, out = complex, real    => irfft
// in, out = real, real       => ERROR
// tmp_storage_size If NULL, library will allocate storage automatically
BFstatus bfFftExecute(BFfft          plan,
                      BFarray const* in,
                      BFarray const* out,
                      BFbool         inverse,
                      void*          tmp_storage,
                      size_t         tmp_storage_size) {
	BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
	BF_ASSERT(in,   BF_STATUS_INVALID_POINTER);
	BF_ASSERT(out,  BF_STATUS_INVALID_POINTER);
	return plan->execute(in, out, inverse, tmp_storage, tmp_storage_size);
}
BFstatus bfFftDestroy(BFfft plan) {
	BF_ASSERT(plan, BF_STATUS_INVALID_HANDLE);
	delete plan;
	return BF_STATUS_SUCCESS;
}
